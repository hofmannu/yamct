#include "hip/hip_runtime.h"
#include "mc.cuh"
#include "mc_kernel.cu"
#include "quaternion_toolbox.cu"

// class constructor
mc::mc()
{
	// generate default material 0 as water
	optProperties water;
	water.set_mua(0.4); // 1/mm
	water.set_mus(2);
	water.set_n(1.31);
	water.set_g(0.9); // does not matter as long as we have no scattering

	tissues.push_back(water);

	fiberProperties startFiber;
	fibers.push_back(startFiber); 
}

// class destructor mostly to free up all the memory
mc::~mc()
{
	// free volumetric data for our heat map
	if (isHeatAlloc)
	{
		delete[] heat;
		delete[] heat_log;
		delete[] fluence;
		delete[] fluence_log;
		// free slices and plots
		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			delete[] slice[iDim];
			delete[] sliceLog[iDim];
			delete[] sliceFluence[iDim];
			delete[] sliceFluenceLog[iDim];
			delete[] vecs[iDim];
			delete[] plotHeat[iDim];
			delete[] plotHeatLog[iDim];
			delete[] plotFluence[iDim];
			delete[] plotFluenceLog[iDim];
		}
	}

	if (isHeatDevAlloc)
	{
		hipFree(heat_dev);
		hipFree(fluence_dev);
	}

}



float* mc::get_slice(
	const uint8_t iDim, // dimension which we cut 
	const float pos, // position where we cut
	const bool flagLog, // flag for log scale
	const bool flagFluence) // flag for fluence scale
{
	// convert position into index
	int32_t idx = (pos - origin[iDim]) / res[iDim];
	
	if (idx < 0) // if below 0, limit to 0
		idx = 0;

	if (idx >= dims[iDim]) // if beyond upper range, also limit
		idx = dims[iDim] - 1;

	// check if idx is same as last time, then simply return, otherwise
	// update slice
	if (idx != idxCross[iDim])
	{
		update_slice_log(iDim, idx);
		update_slice(iDim, idx);
		idxCross[iDim] = idx;
	}

	float* retSlice;

	if (flagFluence)
	{
		if (flagLog)
		{
			retSlice = sliceFluenceLog[iDim];
		}
		else
		{
			retSlice = sliceFluence[iDim];
		}
	}
	else
	{
		if (flagLog)
		{
			retSlice = sliceLog[iDim];
		}
		else
		{
			retSlice = slice[iDim];
		}
	}

	return retSlice;
}

float* mc::get_plot(
	const uint8_t iDim, // dimension along which we request
	const float* pos, // three element vector describing center position
	const bool flagLog,
	const bool flagFluence)
{
	int32_t idx[3];
	bool flagChange = 0;

	// convert position in all three dimensions into an index
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		idx[iDim] = (pos[iDim] - origin[iDim]) / res[iDim];
		if (idx[iDim] < 0)
			idx[iDim] = 0;

		if (idx[iDim] >= dims[iDim])
			idx[iDim] = dims[iDim] - 1;

		if (idx[iDim] != idxCrossPlots[iDim])
			flagChange = 1;
	}

	// if any position changed we update all plots
	if (flagChange)
	{
		update_plots(&idx[0]);
		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			idxCrossPlots[iDim] = idx[iDim];
		}
	}

	float* returnedPlot;

	if (flagFluence) // here we are supposed to return fluence
	{
		if (flagLog) // fluence in log
		{
			returnedPlot = plotFluenceLog[iDim];
		}
		else // fluence in normal
		{
			returnedPlot = plotFluence[iDim];
		}
	}
	else // now we push back with heat
	{
		if (flagLog) // heat in log scale
		{
			returnedPlot = plotHeatLog[iDim];
		}
		else // heat in normal scale
		{
			returnedPlot = plotHeat[iDim];
		}
	}	
	return returnedPlot;
}

// updates all plots along all dimensions
void mc::update_plots(const int32_t* idxPos)
{

	uint32_t idxIn; // iX + nX * (iY * iZ * nY)
	float* currPlot;

	for (uint32_t iX = 0; iX < dims[0]; iX++)
	{
		idxIn = iX + dims[0] * (idxPos[1] + idxPos[2] * dims[1]);
		
		currPlot = plotHeat[0];
		currPlot[iX] = heat[idxIn];

		currPlot = plotHeatLog[0];
		currPlot[iX] = heat_log[idxIn];

		currPlot = plotFluence[0];
		currPlot[iX] = fluence[idxIn];

		currPlot = plotFluenceLog[0];
		currPlot[iX] = fluence_log[idxIn];
	}

	for (uint32_t iY = 0; iY < dims[1]; iY++)
	{
		idxIn = idxPos[0] + dims[0] * (iY + idxPos[2] * dims[1]);

		currPlot = plotHeat[1];
		currPlot[iY] = heat[idxIn];

		currPlot = plotHeatLog[1];
		currPlot[iY] = heat_log[idxIn];

		currPlot = plotFluence[1];
		currPlot[iY] = fluence[idxIn];

		currPlot = plotFluenceLog[1];
		currPlot[iY] = fluence_log[idxIn];
	}

	for (uint32_t iZ = 0; iZ < dims[2]; iZ++)
	{
		idxIn = idxPos[0] + dims[0] * (idxPos[1] + iZ * dims[1]);

		currPlot = plotHeat[2];
		currPlot[iZ] = heat[idxIn];

		currPlot = plotHeatLog[2];
		currPlot[iZ] = heat_log[idxIn];

		currPlot = plotFluence[2];
		currPlot[iZ] = fluence[idxIn];

		currPlot = plotFluenceLog[2];
		currPlot[iZ] = fluence_log[idxIn];
	}

	return;
}

// updates the slice for a given index at a given position
void mc::update_slice(const uint8_t iDim, const uint32_t idx)
{
	float* currSlice = slice[iDim]; // get pointer to slice
	float* currSliceFluence = sliceFluence[iDim];

	uint32_t idxIn, idxOut;

	if (iDim == 0) // slice me along yz
	{
		for (uint32_t iZ = 0; iZ < dims[2]; iZ++)
		{
			for (uint32_t iY = 0; iY < dims[1]; iY++)
			{
				idxIn = idx + iY * dims[0] + iZ * dims[0] * dims[1];
				idxOut = iZ + dims[2] * iY;
				currSlice[idxOut] = heat[idxIn];
				currSliceFluence[idxOut] = fluence[idxIn];
			}
		}
	}
	else if (iDim == 1) // slice me along xz
	{
		for (uint32_t iX = 0; iX < dims[0]; iX++)
		{
			for (uint32_t iZ = 0; iZ < dims[2]; iZ++)
			{
				idxIn =iX + idx * dims[0] + iZ * dims[0] * dims[1];
				idxOut = iZ + dims[2] * iX;
				currSlice[idxOut] = heat[idxIn];
				currSliceFluence[idxOut] = fluence[idxIn];
			}
		}
	}
	else if (iDim == 2) // slice me along xy
	{
		for (uint32_t iX = 0; iX < dims[0]; iX++)
		{
			for (uint32_t iY = 0; iY < dims[1]; iY++)
			{
				idxIn = iX + iY * dims[0] + idx * dims[0] * dims[1];
				idxOut = iX + dims[0] * iY;
				currSlice[idxOut] = heat[idxIn];
				currSliceFluence[idxOut] = fluence[idxIn];
			}
		}
	}
	else
	{
		printf("Invalid dimension order, must be 0, 1, or 3");
		throw "InvalidVal";
	}

	return;
}

// update logarithmic slices
void mc::update_slice_log(const uint8_t iDim, const uint32_t idx)
{
	float* currSlice = sliceLog[iDim]; // get pointer to slice
	float* currSliceFluence = sliceFluenceLog[iDim]; // get pointer to flu slice
	uint32_t idxIn, idxOut;

	if (iDim == 0) // slice me along yz
	{
		for (uint32_t iZ = 0; iZ < dims[2]; iZ++)
		{
			for (uint32_t iY = 0; iY < dims[1]; iY++)
			{
				idxOut = iZ + dims[2] * iY;
				idxIn = idx + iY * dims[0] + iZ * dims[0] * dims[1];
				currSlice[idxOut] = heat_log[idxIn];
				currSliceFluence[idxOut] = fluence_log[idxIn];
			}
		}
	}
	else if (iDim == 1) // slice me along xz
	{
		for (uint32_t iX = 0; iX < dims[0]; iX++)
		{
			for (uint32_t iZ = 0; iZ < dims[2]; iZ++)
			{
				idxOut = iZ + dims[2] * iX;
				idxIn = iX + idx * dims[0] + iZ * dims[0] * dims[1];
				currSlice[idxOut] = heat_log[idxIn];
				currSliceFluence[idxOut] = fluence_log[idxIn];
			}
		}
	}
	else if (iDim == 2) // slice me along xy
	{
		for (uint32_t iX = 0; iX < dims[0]; iX++)
		{
			for (uint32_t iY = 0; iY < dims[1]; iY++)
			{
				idxIn = iX + iY * dims[0] + idx * dims[0] * dims[1];
				idxOut = iX + dims[0] * iY;
				currSlice[idxOut] = heat_log[idxIn];
				currSliceFluence[idxOut] = fluence_log[idxIn];
			}
		}
	}
	else
	{
		printf("Invalid dimension order, must be 0, 1, or 3");
		throw "InvalidVal";
	}

	idxCross[iDim] = idx;

	return;
}

// initialize all required variables and allocate our memory
void mc::init_vars()
{	
	hipSetDevice(sim.get_gpuID());
	hipError_t err;
	// alloc memory for heat map and log version of it on CPU
	if (isHeatAlloc)
	{
		delete[] heat;
		delete[] heat_log;
		delete[] fluence;
		delete[] fluence_log;
		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			delete[] slice[iDim];
			delete[] sliceLog[iDim];
			delete[] vecs[iDim];	
			delete[] plotHeat[iDim];
			delete[] plotHeatLog[iDim];
			delete[] plotFluence[iDim];
			delete[] plotFluenceLog[iDim];
		}
	}

	// allocate memory for heat and fluence on host
	heat = new float [volume.get_nElem() + 1]; // last element is container for overfly
	heat_log = new float [volume.get_nElem()];
	fluence = new float [volume.get_nElem()];
	fluence_log = new float [volume.get_nElem()];

	// grab dimensions etc from volume
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		dims[iDim] = volume.get_dim(iDim);
		res[iDim] = volume.get_res(iDim);
		origin[iDim] = volume.get_min(iDim);
	}

	// allocate memory for vectors and their plots
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		vecs[iDim] = new float [dims[iDim]];
		float* currVec = vecs[iDim];
		for (uint32_t iXYZ = 0; iXYZ < dims[iDim]; iXYZ++)
			currVec[iXYZ] = origin[iDim] + res[iDim] + ((float) iXYZ + 0.5);

		plotHeat[iDim] = new float [dims[iDim]];
		plotHeatLog[iDim] = new float [dims[iDim]];
		plotFluence[iDim] = new float [dims[iDim]];
		plotFluenceLog[iDim] = new float [dims[iDim]];
	}

	// allocate memory for slices
	slice[0] = new float [dims[1] * dims[2]]; // along x
	slice[1] = new float [dims[0] * dims[2]]; // along y
	slice[2] = new float [dims[0] * dims[1]]; // along z
	sliceLog[0] = new float [dims[1] * dims[2]]; // along x
	sliceLog[1] = new float [dims[0] * dims[2]]; // along y
	sliceLog[2] = new float [dims[0] * dims[1]]; // along z

	sliceFluence[0] = new float [dims[1] * dims[2]]; // along x
	sliceFluence[1] = new float [dims[0] * dims[2]]; // along y
	sliceFluence[2] = new float [dims[0] * dims[1]]; // along z
	sliceFluenceLog[0] = new float [dims[1] * dims[2]]; // along x
	sliceFluenceLog[1] = new float [dims[0] * dims[2]]; // along y
	sliceFluenceLog[2] = new float [dims[0] * dims[1]]; // along z

	isHeatAlloc = 1;

	for (uint64_t idx = 0; idx < volume.get_nElem(); idx++)
	{
		heat[idx] = 0;
		heat_log[idx] = 0;
		fluence[idx] = 0;
		fluence_log[idx] = 0;
	}
	heat[volume.get_nElem()] = 0; // set collection bin also to 0

	// allocate memory for our crosssection through the heat map

	// allocate matrix for heat map on GPU
	if (isHeatDevAlloc)
		hipFree(heat_dev);

	// allocate memory on device for absorption map
	err = hipMalloc( (void**)&heat_dev, (volume.get_nElem() + 1) * sizeof(float) );
	if (err != hipSuccess)
	{
		printf("[mc] Could not allocate required memory for heat on card\n");
		printf("[mc] Size of requested array: nElements = %d\n", 
			volume.get_nElem());
		printf(hipGetErrorString(err));
		printf("\n");
		throw "MemoryAllocError";
	}

	isHeatDevAlloc = 1;

	// copy zero vectors over for fluence and heat
	bool cpy1 = (hipSuccess != hipMemcpy(heat_dev, heat, 
		(volume.get_nElem() + 1) * sizeof(float), hipMemcpyHostToDevice));

	if (cpy1){
		printf("Could not copy memory to card\n");	
		throw "MemoryCopyError";
	}

	return;
}

// starts the actual simulation
void mc::run_sim()
{
	hipSetDevice(sim.get_gpuID());

	clock_t begin = clock(); // start stopwatch
	hipError_t err; // variable used to handle CUDA errors

		// define constant simulation properties for our field	
	constArgsIn inArgs;
	for (uint8_t iDim = 0; iDim < 3; iDim++)
	{
		inArgs.dim[iDim] = volume.get_dim(iDim);
		inArgs.res[iDim] =  volume.get_res(iDim);
		inArgs.origin[iDim] = volume.get_min(iDim);
		inArgs.maxPos[iDim] = volume.get_max(iDim);
	}
	inArgs.bgMaterial = volume.get_bgMaterialId();
	inArgs.killFlag = sim.get_flagKillBound();
	inArgs.nElements = inArgs.dim[0] * inArgs.dim[1] * inArgs.dim[2];

	// define optical properties of our materials
	int nMaterial = tissues.size();
	optProps* optPropHost = new optProps[nMaterial];
	for (uint8_t iMaterial = 0; iMaterial < nMaterial; iMaterial++)
	{
		// make sure that there is no zero absorption
		if (tissues[iMaterial].get_mua() < 1e-5)
			tissues[iMaterial].set_mua(1e-5);

		optPropHost[iMaterial].mu_a = tissues[iMaterial].get_mua();
		optPropHost[iMaterial].mu_s = tissues[iMaterial].get_mus();
		optPropHost[iMaterial].g = tissues[iMaterial].get_g();

		// calculated values
		optPropHost[iMaterial].albedo = tissues[iMaterial].get_albedo();
		optPropHost[iMaterial].albedo1 = 1 - optPropHost[iMaterial].albedo;
		optPropHost[iMaterial].mu_as = 1 / (optPropHost[iMaterial].mu_a + optPropHost[iMaterial].mu_s);
		optPropHost[iMaterial].g2 = optPropHost[iMaterial].g * optPropHost[iMaterial].g;
		optPropHost[iMaterial].gx2 = 2.0 * optPropHost[iMaterial].g;
	}

	// allocate memory for fiber properties
	int nFibers = fibers.size(); // number of fibers
	fiberProps* fiberPropHost = new fiberProps[nFibers];

	float totalWeight = 0;
	for (uint8_t iFiber = 0; iFiber < nFibers; iFiber++)
		totalWeight += fibers[iFiber].get_weight();
	
	float lastWeight = 0;
	for (uint8_t iFiber = 0; iFiber < nFibers; iFiber++)
	{
		// normalize fiber orientation
		const float normOr = sqrt(
			fibers[iFiber].get_orientation(0) * fibers[iFiber].get_orientation(0) +
			fibers[iFiber].get_orientation(1) * fibers[iFiber].get_orientation(1) +
			fibers[iFiber].get_orientation(2) * fibers[iFiber].get_orientation(2));

		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			fiberPropHost[iFiber].pos[iDim] = fibers[iFiber].get_pos(iDim);
			fiberPropHost[iFiber].orientation[iDim] = 
				fibers[iFiber].get_orientation(iDim) / normOr;
		}
		fiberPropHost[iFiber].rCore = fibers[iFiber].get_rCore();
		fiberPropHost[iFiber].rCore2  = fiberPropHost[iFiber].rCore * 
			fiberPropHost[iFiber].rCore;
		fiberPropHost[iFiber].na = fibers[iFiber].get_numAp();
		
		// TODO replace this with the actual calculation depending on the medium
		fiberPropHost[iFiber].phiMax = fibers[iFiber].get_theta(1.33);
		
		// do decide which fiber emmits photon package we generate a random number
		// between 0 and 1 and project it onto a rated fiber range
		lastWeight += fibers[iFiber].get_weight() / totalWeight;
		fiberPropHost[iFiber].randMax = lastWeight;
	}
	fiberPropHost[nFibers - 1].randMax = 1.001;

	// allocate memory for tissue mapping
	uint8_t* tissueTypes_dev;
	err = hipMalloc( (void**)&tissueTypes_dev, volume.get_nElem() * sizeof(uint8_t) );
	if (err != hipSuccess)
	{
		printf("Could not allocate memory on card for tissue type map.\n");
		throw "cudaMallocErr";
	}

	hipMemcpy(tissueTypes_dev, volume.get_pvolumeData(), volume.get_nElem() * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("Could not copy settings struct over to GPU.\n");
		throw "cudaMemcpyErr";
	}	

	// allocate memory on GPU for constant arguments and copy them over
	constArgsIn* inArgs_dev;
	err = hipMalloc( (void**)&inArgs_dev, sizeof(constArgsIn) );
	if (err != hipSuccess)
	{
		printf("Could not allocate memory on card for settings struct.\n");
		throw "cudaMallocErr";
	}

	hipMemcpy(inArgs_dev, &inArgs, sizeof(constArgsIn), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("Could not copy settings struct over to GPU.\n");
		throw "cudaMemcpyErr";
	}

	// allocate memory on GPU for tissue properties and copy them over
	optProps* optProp_dev;
	err = hipMalloc( (void**)&optProp_dev, nMaterial * sizeof(optProps));
	if (err != hipSuccess)
	{
		printf("Could not allocate memory on card for tissue struct.\n");
		throw "cudaMallocErr";
	}

	hipMemcpy(optProp_dev, optPropHost, nMaterial * sizeof(optProps), hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("Could not copy tissue struct over to GPU.\n");
		throw "cudaMemcpyErr";
	}

	// alocate memory on GPU for fiber properties and copy them over
	fiberProps* fiberProp_dev;
	err = hipMalloc( (void**)&fiberProp_dev, sizeof(fiberProps) * nFibers);
	if (err != hipSuccess)
	{
		printf("Could not allocate memory on card for fiber struct.\n");
		throw "cudaMallocErr";
	}

	hipMemcpy(fiberProp_dev, fiberPropHost, sizeof(fiberProps) * nFibers, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		printf("Could not copy fiber struct over to GPU.\n");
		throw "cudaMemcpyErr";
	}

 	// check occupancy before starting

  // These variables are used to convert occupancy to warps
  // hipDeviceProp_t prop;
  // hipGetDeviceProperties(&prop, sim.get_gpuID());
    
  // hipOccupancyMaxActiveBlocksPerMultiprocessor(
  // 	&numBlocks,
  //   simPhoton,
  //   sim.get_threadsPerBlock(),
  //   0);

  int blockSize;      // The launch configurator returned block size
  int minGridSize;    // The minimum grid size needed to achieve the
          
  hipOccupancyMaxPotentialBlockSize(
      &minGridSize,
      &blockSize,
      (void*) simPhoton,
      0,
      sim.get_nPhotonsTrue());

  int gridSize = ((sim.get_nPhotonsTrue()) + blockSize - 1) / blockSize;

	// start actual simulation
	printf("Starting actual simulation with %d blocks, each %d threads\n",
		gridSize, blockSize);

	simPhoton<<<gridSize, blockSize >>>(
		heat_dev, // output matrix into which we write our absorption
		inArgs_dev, // constant simulation parameters
		tissueTypes_dev, // defines the distribution of our tissue types
		optProp_dev, // random number 
		fiberProp_dev); // fiber properties
	hipDeviceSynchronize();	

	err = hipGetLastError();
	if (err != hipSuccess){
		printf("Error occured during simulation: ");
		printf(hipGetErrorString(err));
	}else{	

		// copy heat array back from gpu
		err = hipMemcpy(heat, heat_dev, 
			(volume.get_nElem() + 1) * sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			printf("Could not copy heat array back from GPU\n");
			throw "MemoryCopyError";
		}

		clock_t end = clock();
	
		simTime = (end - begin) / (double) CLOCKS_PER_SEC;
		//printf("Time elapsed in [s]: %f \n", simTime);
		//printf("Time per photon package in [ns]: %f \n", simTime / (double) sim.get_nPhotonsTrue() * 1e9);
		//printf("Photon packages per ms: %f \n", (double) sim.get_nPhotonsTrue() / (simTime * 1000));	
	}
	
	// free host and device memory
	hipFree(heat_dev); // free device memorz for heat_dev
	hipFree(fluence_dev);
	isHeatDevAlloc = 0;

	hipFree(optProp_dev);
	hipFree(fiberProp_dev);
	delete[] optPropHost;
	delete[] fiberPropHost;

	hipFree(inArgs_dev);

	// for debugging: calculate and print sum of heat
	float sum = 0;
	uint32_t nanCount = 0;
	for (uint32_t iElem = 0; iElem <= volume.get_nElem(); iElem++)
	{
		if (!isnan(heat[iElem]))
			sum += heat[iElem];
		else
			nanCount++;
	}
	photonRatio = sum / ((float) sim.get_nPhotonsTrue());
	if (flagDebug)
	{
		printf("[debug] Overall photons found (0 ... 1): %.2f, nans: %d\n", 
			photonRatio, nanCount);
		printf("[debug] Percentage in collection bin (0 ... 1): %.2f\n", 
			heat[volume.get_nElem()] / sum);
	}

	// scale heat by volume and nphotons
	const float scalingFac = (float) sim.get_nPhotonsTrue() * volume.get_volume_voxel();
	uint8_t* volumeData = volume.get_pvolumeData();
	for (uint32_t iElem = 0; iElem < volume.get_nElem(); iElem++)
	{
		heat[iElem] /= scalingFac;
		const float muaTemp = tissues[volumeData[iElem]].get_mua();
		fluence[iElem] = heat[iElem] / muaTemp;
	}
	heat[volume.get_nElem()] /= scalingFac;

	calcMinMax(); // calculate minimum and maximum value in heat map
	if (flagDebug)
	{
	 	printf("[debug] maximum value in field: %f, minimum value in field: %f\n", 
	 		maxVal, minVal);
	}
	
	calcLog(); // calculate logarthmic represntation of field
	for (uint8_t iDim = 0; iDim < 3; iDim++) // generate an initial set of slices
	{
		update_slice(iDim, 0);
		update_slice_log(iDim, 0);	
	}
	
	return;
}

// claulates the minimum and the maximum value in our volume both for log and linear
// scale
void mc::calcMinMax()
{
	// get maximum and minimum value in our mip
	minVal = heat[0];
	maxVal = heat[0];
	minFluence = fluence[0];
	maxFluence = fluence[0];

	minValLog = 0; // log10(heat[0])
	maxValLog = 0; // log10(heat[0])
	minFluenceLog = 0;
	maxFluenceLog = 0;

	for (uint64_t iElement = 1; iElement < volume.get_nElem(); iElement++)
	{

		// check if we have a new maximum for our heat vector
		if (heat[iElement] > maxVal)
		{
			maxVal = heat[iElement];
			if (maxVal > 0)
			{
				maxValLog = log10(maxVal);
			}
		}

		// check if we have a new maximum for our fluence vector
		if (fluence[iElement] > maxFluence)
		{
			maxFluence = fluence[iElement];
			if (maxFluence > 0)
			{
				maxFluenceLog = log10(fluence[iElement]);
			}
		}

		// check if we have a new minimum for our heat vector
		if (heat[iElement] < minVal)
		{
			minVal = heat[iElement];
			// only update our log min if we are bigger then 0
			if (minVal > 0)
			{
				minValLog = log10(minVal);
			}
		}

		// check if we have a new minimum for our heat vector
		if (fluence[iElement] < minFluence)
		{
			minFluence = fluence[iElement];
			// only update our log min if we are bigger then 0
			if (minFluence > 0)
			{
				minFluenceLog = log10(minFluence);
			}
		}
	}

	return;
}

// converts the calculated heat map into logarthmic scale
void mc::calcLog()
{
	for (uint64_t iElement = 0; iElement < volume.get_nElem(); iElement++)
	{
		// calc logarithmic value for heat
		if (heat[iElement] > 0)
			heat_log[iElement] = log10(heat[iElement]);
		else
			heat_log[iElement] = -38; // lower end of float accuracy

		// calc logartihmic value for fluence
		if (fluence[iElement] > 0)
			fluence_log[iElement] = log10(fluence[iElement]);
		else
			fluence_log[iElement] = -38;
	}
	return;
}

void mc::run()
{
	// calc_reflectance(nWater, tissue.get_n());
	isDone = 0;
	init_vars();
	run_sim();
	isDone = 1;
	return;
}

// exports heat map as a vtk file
bool mc::exportVtk(const string filePath)
{
	bool success = 1;
	try{
		vtkwriter myWriter;

		const string title ("fluence");
		myWriter.set_title(title);
		const string type ("STRUCTURED_POINTS");
		myWriter.set_type(type);
		myWriter.set_outputPath(filePath);

		griddedData myData;

		// todo needs fixing here
		myData.data = heat;

		for (uint8_t iDim = 0; iDim < 3; iDim++)
		{
			myData.origin[iDim] = origin[iDim]; // origin in z
			myData.res[iDim] = res[iDim]; // resolution in z direction
			myData.dim[iDim] = dims[iDim];
		}

		myWriter.set_structuredPoints(&myData);
		myWriter.set_binary();

		if (flagDebug)
			printf("[debug] Writing data to file\n");

		myWriter.write();
	}
	catch(...)
	{
		printf("Could absolutely not save dataset to file\n");
	}

	return success;
}

// save fluence, heat, dimensions, resolution and origin as h5 
bool mc::exportH5(const string filePath)
{
	bool success = 1;
	try{

		H5::H5File file(filePath, H5F_ACC_TRUNC);

		// write resolutiion to file
		//printf("Write resolution to file...\n");
		const hsize_t col_dims = 3;
		H5::DataSpace mspaceRes(1, &col_dims);
		H5::DataSet resDataset = file.createDataSet(
			"dr", H5::PredType::NATIVE_FLOAT, mspaceRes);
		resDataset.write(res, H5::PredType::NATIVE_FLOAT);
		resDataset.close();

		// write origin to file
		//printf("Write origin to file...\n");
		H5::DataSpace mspaceOrigin(1, &col_dims);
		H5::DataSet originDataset = file.createDataSet(
			"origin", H5::PredType::NATIVE_FLOAT, mspaceOrigin);
		originDataset.write(origin, H5::PredType::NATIVE_FLOAT);
		originDataset.close();

		// write dimension to file
		//printf("Write dimensions to file...\n");
		H5::DataSpace mspaceDim(1, &col_dims);
		H5::DataSet dimDataset = file.createDataSet(
			"dim", H5::PredType::NATIVE_UINT, mspaceDim);
		dimDataset.write(dims, H5::PredType::NATIVE_UINT32);
		dimDataset.close();

		// write heat map to file
		const hsize_t col_data = dims[0] * dims[1] * dims[2];
		H5::DataSpace mspaceData(1, &col_data);
		H5::DataSet dataDataset = file.createDataSet(
			"heat", H5::PredType::NATIVE_FLOAT, mspaceData);
		dataDataset.write(heat, H5::PredType::NATIVE_FLOAT);
		dataDataset.close();

		// write fluence map to file
		H5::DataSpace mspaceFluence(1, &col_data);
		H5::DataSet fluenceDataset = file.createDataSet(
			"fluence", H5::PredType::NATIVE_FLOAT, mspaceFluence);
		fluenceDataset.write(fluence, H5::PredType::NATIVE_FLOAT);
		fluenceDataset.close();

		file.close();
	}
	catch(...)
	{
		printf("Could absolutely not save data to file\n");
		success = 0;
	}

	return success;
}

// save dataset as h5 to default path
bool mc::exportH5()
{
	string filePath = "default.h5";
	bool success = exportH5(filePath);
	return success;
}

// writes all the settings of the reconstruction procedure to a file
void mc::write_settings(const string filePath)
{
	json j;

	// simulation properties
	j["nPhotons"] = sim.get_nPhotons();
	j["killFlag"] = sim.get_flagKillBound();
	j["nFibers"] = fibers.size();
	j["nTissues"] = tissues.size();

	// fiber properties
	for (uint8_t iFiber = 0; iFiber < fibers.size(); iFiber++)
	{
		char nameCurr[80];
		sprintf(nameCurr, "fiber%d", iFiber);
		j[nameCurr] = {
			{"numAp", fibers[iFiber].get_numAp()},
			{"dCore", fibers[iFiber].get_dCore()},
			{"posX", fibers[iFiber].get_pos(0)},
			{"posY", fibers[iFiber].get_pos(1)},
			{"posZ", fibers[iFiber].get_pos(2)},
			{"dirX", fibers[iFiber].get_orientation(0)},
			{"dirY", fibers[iFiber].get_orientation(1)},
			{"dirZ", fibers[iFiber].get_orientation(2)},
			{"weight", fibers[iFiber].get_weight()}
		};
	}

	// tissue properties
	for (uint8_t iTissue = 0; iTissue < tissues.size(); iTissue++)
	{
		char nameCurr[80];
		sprintf(nameCurr, "tissue%d", iTissue);
		j[nameCurr] = {
			{"mua", tissues[iTissue].get_mua()},
			{"mus", tissues[iTissue].get_mus()},
			{"g", tissues[iTissue].get_g()},
			{"n", tissues[iTissue].get_n()}
		};
	}

	json jGeom = volume.get_json();
	j.insert(jGeom.begin(), jGeom.end());

	std::ofstream o(filePath);
	o << j << endl;

	return;
}

// read settings back from the json file
void mc::read_settings(const string filePath)
{
	ifstream ifs(filePath);
	json inputJs = json::parse(ifs);

	sim.set_nPhotons(inputJs["nPhotons"]);
	sim.set_flagKillBound(inputJs["killFlag"]);

	uint8_t nFibers = inputJs["nFibers"];
	uint8_t nTissues = inputJs["nTissues"];

	// empty all currently existing fibers
	fibers.clear();

	// load all fiber definitions from file
	for (uint8_t iFiber = 0; iFiber < nFibers; iFiber++)
	{
		char nameCurr[80];
		sprintf(nameCurr, "fiber%d", iFiber);
		fiberProperties newFiber;
		newFiber.set_numAp(inputJs[nameCurr]["numAp"]);
		newFiber.set_dCore(inputJs[nameCurr]["dCore"]);
		newFiber.set_pos(0, inputJs[nameCurr]["posX"]);
		newFiber.set_pos(1, inputJs[nameCurr]["posY"]);
		newFiber.set_pos(2, inputJs[nameCurr]["posZ"]);
		newFiber.set_orientation(0, inputJs[nameCurr]["dirX"]);
		newFiber.set_orientation(1, inputJs[nameCurr]["dirY"]);
		newFiber.set_orientation(2, inputJs[nameCurr]["dirZ"]);
		newFiber.set_weight(inputJs[nameCurr]["weight"]);
		fibers.push_back(newFiber);
	} 

	// read optical properties from settings file
	tissues.clear();
	for (uint8_t iTissue = 0; iTissue < nTissues; iTissue++)
	{
		char nameCurr[80];
		sprintf(nameCurr, "tissue%d", iTissue);
		optProperties newProp;
		newProp.set_mua(inputJs[nameCurr]["mua"]);
		newProp.set_mus(inputJs[nameCurr]["mus"]);
		newProp.set_g(inputJs[nameCurr]["g"]);
		newProp.set_n(inputJs[nameCurr]["n"]);

		tissues.push_back(newProp);
	}

	// read geometry from file
	volume.read_json(inputJs);

	return;
}
